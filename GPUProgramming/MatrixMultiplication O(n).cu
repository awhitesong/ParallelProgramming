#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include<iostream>
#include<stdlib.h>
#include<assert.h>
using namespace std;
void initCUDA(int*, int*, int*, int);

__global__ void multiply_(int *c, int *a, int *b, int n){
	size_t row, col;
	int k,temp=0;
	row = threadIdx.x;
	col = threadIdx.y;
	for (k = 0; k < n; k++){
		temp += a[row*n + k] * b[k*n + col];
	}
	c[row*n + col] = temp;
}

main(){
	int i, j, n, *a, *b, *c, k, *gpu_result;
	cout << "Enter the square matrix's dimensions (maximum 32)\n";
	cin >> n;
	assert(n <= 32);
	a = (int*)malloc(sizeof(int)*n*n);
	b = (int*)malloc(sizeof(int)*n*n);
	gpu_result = (int*)malloc(sizeof(int)*n*n);
	c = (int*)calloc(n*n, sizeof(int));
	cout << "Enter Matrice A: \n\n";
	for (i = 0; i < n; i++){
		for (j = 0; j < n; j++){
			cin >> a[i*n + j];
		}
	}
	cout << "\nEnter Matrice B: \n\n";
	for (i = 0; i < n; i++){
		for (j = 0; j < n; j++){
			cin >> b[i*n + j];
		}
	}
	cout << "\nCPU Multiplication in O(n^3):\n\n";
	for (i = 0; i < n; i++){
		for (j = 0; j < n; j++){
			for (k = 0; k < n; k++){
				c[i*n + j] += a[i*n + k] * b[k*n + j];
			}
		}
	}
	for (i = 0; i < n; i++){
		for (j = 0; j < n; j++){
			cout << c[i*n + j] << " ";
		}
		cout << endl;
	}
	cout << endl;
	initCUDA(a, b, gpu_result, n);

	cout << "GPU Multiplication in O(n):\n\n";

	for (i = 0; i < n; i++){
		for (j = 0; j < n; j++){
			cout << gpu_result[i*n + j] << " ";
		}
		cout << endl;
	}

	free(a);
	free(b);
	free(c);
	free(gpu_result);
}


void initCUDA(int *a, int *b, int *result, int n){
	int *d_a, *d_b, *d_c;
	hipMalloc(&d_a, n*n*sizeof(int));
	hipMalloc(&d_b, n*n*sizeof(int));
	hipMalloc(&d_c, n*n*sizeof(int));
	hipMemcpy(d_a, a, n*n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, n*n*sizeof(int), hipMemcpyHostToDevice);
	dim3 threadsinablock(n, n);
	dim3 blocksingrid(1, 1);
	multiply_ <<<blocksingrid, threadsinablock >>>(d_c, d_a, d_b, n);
	hipMemcpy(result, d_c, n*n*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

}